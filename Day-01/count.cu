#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int count;
    hipError_t err = hipGetDeviceCount(&count);

    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }

    std::cout << "Number of CUDA devices: " << count << std::endl;
    return 0;
}
