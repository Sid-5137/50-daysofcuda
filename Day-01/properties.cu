#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void){
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    printf( "   --- General Information ---\n");
    printf("Name: %d\n",prop.name);
    printf("Compute Capability: %d.%d\n",prop.major,prop.minor);
    printf("Clock Rate: %d\n",prop.clockRate);
    printf( "Device copy overlap:" );
    if (prop.deviceOverlap)
        printf( "Enabled\n" );
    else
        printf( "Disabled\n" );
    printf("Kernel execition timeout : " );
    if (prop.kernelExecTimeoutEnabled)
        printf( "Enabled\n" );
    else
        printf( "Disabled\n" );
    printf( "   --- Memory Information for device ---\n");
    printf( "Total global mem: %ld\n", prop.totalGlobalMem );
    printf( "Total constant Mem: %ld\n", prop.totalConstMem );
    printf( "Max mem pitch: %ld\n", prop.memPitch );
    printf( "Texture Alignment: %ld\n", prop.textureAlignment );
    printf( "   --- MP Information for device ---\n");
    printf( "Multiprocessor count: %d\n", prop.multiProcessorCount );
    printf( "Shared mem per mp: %ld\n", prop.sharedMemPerBlock );
    printf( "Registers per mp: %d\n", prop.regsPerBlock );
    printf( "Threads in warp: %d\n", prop.warpSize );
    printf( "Max threads per block: %d\n", prop.maxThreadsPerBlock );
    printf( "Max thread dimensions:(%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2] );
    printf( "Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2] );
    printf( "\n" );

    return 0;
}